#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include "typedefs.h"
#include ""

#include <thrust\scan.h>
#include <thrust\device_ptr.h>

#include <cub/device/device_scan.cuh>

#include "util.hcu"
#include "cutil_math.h"
#include "tables.h"
#include "GPUTimer.h"
#include "cuda_utilities.h"

using namespace std;

extern "C" {

	__constant__ uint d_edgeTable[EDGE_SIZE];
	__constant__ uint d_triTable[TRI_ROWS][TRI_COLS];
	__constant__ uint d_countTable[TRI_ROWS];

	const uint MAX_TRIANGLES = 15;


	void allocateTables() {
		hipMemcpyToSymbol(HIP_SYMBOL(d_edgeTable), edgeTable, sizeof(edgeTable));
		hipMemcpyToSymbol(HIP_SYMBOL(d_triTable), triTable, sizeof(triTable));
		hipMemcpyToSymbol(HIP_SYMBOL(d_countTable), numVertsTable, sizeof(numVertsTable));
	}

	__device__ 
		inline float3 cornerValue(const uint3 co, const float3 minX, const float3 dx) {
			return make_float3(minX.x + co.x*dx.x, minX.y + co.y*dx.y, minX.z + co.z*dx.z); 
	}

	__device__ 
		inline float func(float3 co) {
			return co.x*co.x + co.y*co.y + co.z*co.z - 1;
	}

	__device__ 
		inline void interpValues(float isoValue, const float v0, const float v1, float3 p0, float3 p1, float3& out) {
			float mu = (isoValue - v0) / (v1 - v0);
			out = lerp(p0, p1, mu);
	}

	__device__
		inline uint getEdge(uint i) {
			return edgeTable[i];
	}

	__global__ 
		void countKernel(float isoValue, dim3 dims, float3 minX, float3 dx, uint* count, uint* isOccupied, uint N) {
			uint idx = blockIdx.x*blockDim.x + threadIdx.x;

			if (idx >= N) return;


			uint3 co = idx_to_co(idx, dims);

			float3 corners[8];
			corners[0] = cornerValue(co, minX, dx);
			corners[1] = corners[0] + make_float3(dx.x, 0,    0);
			corners[2] = corners[0] + make_float3(dx.x, dx.y, 0);
			corners[3] = corners[0] + make_float3(0,    dx.y, 0);

			corners[4] = corners[0] + make_float3(0,    0,    dx.z);
			corners[5] = corners[0] + make_float3(dx.x, 0,    dx.z);
			corners[6] = corners[0] + make_float3(dx.x, dx.y, dx.z);
			corners[7] = corners[0] + make_float3(0,    dx.y, dx.z);

//			float value[8];
//#pragma unroll 8
//			for (int i = 0; i < 8; i++) {
//				value[i] = func(corners[i]);
//			}

			uint cubeindex;
			cubeindex =  uint(func(corners[0]) < isoValue); 
			cubeindex += uint(func(corners[1]) < isoValue)*2; 
			cubeindex += uint(func(corners[2]) < isoValue)*4; 
			cubeindex += uint(func(corners[3]) < isoValue)*8; 
			cubeindex += uint(func(corners[4]) < isoValue)*16; 
			cubeindex += uint(func(corners[5]) < isoValue)*32; 
			cubeindex += uint(func(corners[6]) < isoValue)*64; 
			cubeindex += uint(func(corners[7]) < isoValue)*128;

			uint nVertices = d_countTable[cubeindex];
			count[idx] = nVertices;
			isOccupied[idx] = nVertices > 0;
	}

	__global__
		void compact(uint* isOccupied, uint* occupiedScan, uint* occupiedCompact) {
			uint idx = blockIdx.x*blockDim.x + threadIdx.x;
			if (isOccupied[idx]) {
				occupiedCompact[occupiedScan[idx]] = idx;
			}
	}

	__global__
		void fillTriangles(float isoValue, dim3 dims, float3 minX, float3 dx, float3* out, uint* vertexPrefix, uint N) {
			uint idx = blockIdx.x*blockDim.x + threadIdx.x;

			if (idx >= N) return;

			uint3 co = idx_to_co(idx, dims);

			float3 corners[8];
			corners[0] = cornerValue(co, minX, dx);
			corners[1] = corners[0] + make_float3(dx.x, 0,    0);
			corners[2] = corners[0] + make_float3(dx.x, dx.y, 0);
			corners[3] = corners[0] + make_float3(0,    dx.y, 0);

			corners[4] = corners[0] + make_float3(0,    0,    dx.z);
			corners[5] = corners[0] + make_float3(dx.x, 0,    dx.z);
			corners[6] = corners[0] + make_float3(dx.x, dx.y, dx.z);
			corners[7] = corners[0] + make_float3(0,    dx.y, dx.z);

			float value[8];

#pragma unroll 8
			for (int i = 0; i < 8; i++) {
				value[i] = func(corners[i]);
			}

			uint cubeindex;
			cubeindex =  uint(value[0] < isoValue); 
			cubeindex += uint(value[1] < isoValue)*2; 
			cubeindex += uint(value[2] < isoValue)*4; 
			cubeindex += uint(value[3] < isoValue)*8; 
			cubeindex += uint(value[4] < isoValue)*16; 
			cubeindex += uint(value[5] < isoValue)*32; 
			cubeindex += uint(value[6] < isoValue)*64; 
			cubeindex += uint(value[7] < isoValue)*128;



			float3 vertList[12];

			if (getEdge(cubeindex) & 1)
				interpValues(isoValue,value[0],value[1],corners[0],corners[1], vertList[0]);
			if (getEdge(cubeindex) & 2)
				interpValues(isoValue,value[1],value[2],corners[1],corners[2], vertList[1]);
			if (getEdge(cubeindex) & 4)
				interpValues(isoValue,value[2],value[3],corners[2],corners[3], vertList[2]);
			if (getEdge(cubeindex) & 8)
				interpValues(isoValue,value[3],value[0],corners[3],corners[0], vertList[3]);
			if (getEdge(cubeindex) & 16)
				interpValues(isoValue,value[4],value[5],corners[4],corners[5], vertList[4]);
			if (getEdge(cubeindex) & 32)
				interpValues(isoValue,value[5],value[6],corners[5],corners[6], vertList[5]);
			if (getEdge(cubeindex) & 64)
				interpValues(isoValue,value[6],value[7],corners[6],corners[7], vertList[6]);
			if (getEdge(cubeindex) & 128)
				interpValues(isoValue,value[7],value[4],corners[7],corners[4], vertList[7]);
			if (getEdge(cubeindex) & 256)
				interpValues(isoValue,value[0],value[4],corners[0],corners[4], vertList[8]);
			if (getEdge(cubeindex) & 512)
				interpValues(isoValue,value[1],value[5],corners[1],corners[5], vertList[9]);
			if (getEdge(cubeindex) & 1024)
				interpValues(isoValue,value[2],value[6],corners[2],corners[6], vertList[10]);
			if (getEdge(cubeindex) & 2048)
				interpValues(isoValue,value[3],value[7],corners[3],corners[7], vertList[11]);

			
			const uint offset = vertexPrefix[idx];

			for (uint i = 0; i < MAX_TRIANGLES; i++) {
				uint edge = d_triTable[cubeindex][i];
				if (edge == 255) break;

				out[offset + i] = vertList[edge];
			}
	}

	void exclusiveScan(uint* in, uint* out, uint N) {
		/*thrust::exclusive_scan(thrust::device_ptr<unsigned int>(in),
			thrust::device_ptr<unsigned int>(in + N),
			thrust::device_ptr<unsigned int>(out));*/

		using namespace hipcub;
		void *d_temp_storage = NULL;
		size_t temp_storage_bytes = 0;
		DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, in, out, N);
		// Allocate temporary storage for exclusive prefix sum
		hipMalloc(&d_temp_storage, temp_storage_bytes);
		// Run exclusive prefix sum
		DeviceScan::ExclusiveSum(d_temp_storage, temp_storage_bytes, in, out, N);

		hipFree(d_temp_storage);
	}

	uint retrieve(uint* array, uint element) {
		uint result;
		hipMemcpy(&result, (array + element), sizeof(uint), hipMemcpyDeviceToHost);
		return result;
	}

	int main() {
		using namespace Gadgetron;
		GPUTimer* t;

		t = new GPUTimer("Const alloc");
		allocateTables();
		delete t;

		
		//hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		hipDeviceSetCacheConfig(hipFuncCachePreferShared);
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(countKernel), hipFuncCachePreferL1);
		hipFuncSetCacheConfig(reinterpret_cast<const void*>(fillTriangles), hipFuncCachePreferL1);
		int n = 100;
		uint3 dims = make_uint3(1, 1, 1) * n;
		float3 min = make_float3(1, 1, 1)*-1.2f;
		float3 dx = make_float3(1, 1, 1)*0.02f;

		const uint N = prod(dims);
		uint* d_count;
		uint *d_occupied, *d_occupiedCompact, *d_occupiedScan;
		float3* d_pos;


		//t = new GPUTimer("Malloc");
		hipMalloc((void **) &d_count, (N+1)*sizeof(uint));
		hipMalloc((void **) &d_occupied, (N+1)*sizeof(uint));
		hipMalloc((void **) &d_occupiedScan, (N+1)*sizeof(uint));
		//delete t;

		//t = new GPUTimer("Running kernel");
		hipDeviceSynchronize();
		t = new GPUTimer("Total");

		{
		int blockSize = 12*32;
		int nBlocks = N/blockSize + (N%blockSize != 0);
		countKernel <<< nBlocks, blockSize >>> (0, dims, min, dx, d_count, d_occupied, N);
		}
		//hipDeviceSynchronize();
		//delete t;
		//CHECK_FOR_CUDA_ERROR();

		//hipDeviceSetCacheConfig(hipFuncCachePreferShared);
		
		//t = new GPUTimer("Scan occupied");
		exclusiveScan(d_occupied, d_occupiedScan, N+1);
		//delete t;

		//t = new GPUTimer("Transfer last occupied element");
		uint nVoxel = retrieve(d_occupiedScan, N);
		//cout << nVoxel << endl;
		//delete t;

		//t = new GPUTimer("Malloc compact");
		hipMalloc((void **) &d_occupiedCompact, nVoxel*sizeof(uint));
		//delete t;

		//t = new GPUTimer("Compact");
		{
		int blockSize = 1*n;
		int nBlocks = N/blockSize + (N%blockSize != 0);
		compact <<< nBlocks, blockSize >>> (d_occupied, d_occupiedScan, d_occupiedCompact);
		}
		//delete t;

		//t = new GPUTimer("Scan count");
		exclusiveScan(d_count, d_count, N+1);
		//delete t;

		//t = new GPUTimer("Transfer last scan element");
		uint nVertex = retrieve(d_count, N);
		//cout << nVertex << endl;
		//delete t;
		//CHECK_FOR_CUDA_ERROR();

		//t = new GPUTimer("Alloc vertex array");
		hipMalloc((void **) &d_pos, nVertex*sizeof(float3));
		//delete t;
		//CHECK_FOR_CUDA_ERROR();

		//hipDeviceSetCacheConfig(hipFuncCachePreferL1);
		//t = new GPUTimer("Gen triangles");
		{
		int blockSize = 5*32;
		int nBlocks = nVoxel/blockSize + (nVoxel%blockSize != 0);
		fillTriangles <<< nBlocks, blockSize >>> (0, dims, min, dx, d_pos, d_count, nVoxel);
		}
		//delete t;
		//CHECK_FOR_CUDA_ERROR();


		float3* h_pos = new float3[nVertex];
		//t = new GPUTimer("Memcpy");
		hipMemcpy(h_pos, d_pos, nVertex * sizeof(float3), hipMemcpyDeviceToHost);
		delete t;
		//CHECK_FOR_CUDA_ERROR();

		return 0;
		
	}
}
